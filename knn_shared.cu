#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <time.h>

#define CUDA_CALL(x)                                                           \
  {                                                                            \
    if ((x) != hipSuccess) {                                                  \
      printf("CUDA error at %s:%d\n", __FILE__, __LINE__);                     \
      printf("  %s\n", hipGetErrorString(hipGetLastError()));                \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

int d = 1 << 5;

__global__ void generatekey(float *Cx, float *Cy, float *Cz, int lenghtC,
                            int *keys, int d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int digit1 = Cx[i] * d;
  int digit2 = Cy[i] * d;
  int digit3 = Cz[i] * d;
  keys[i] = d * d * digit1 + d * digit2 + digit3;
}

__global__ void findCellStartends(int *keys, int len, int *starts, int *ends) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int support[];
  support[threadIdx.x] = keys[i];
  if (threadIdx.x == blockDim.x - 1) {
    support[blockDim.x] = keys[i + 1];
  }

  if (threadIdx.x == 0) {
    if (i > 0) {
      support[blockDim.x + 1] = keys[i - 1];
    }
  }
  __syncthreads();

  if (i > 0) {
    if (threadIdx.x > 0) {
      if (support[threadIdx.x] != support[threadIdx.x - 1]) {
        starts[support[threadIdx.x]] = i;
      }
    } else {
      if (support[threadIdx.x] != support[blockDim.x + 1]) {
        starts[support[threadIdx.x]] = i;
      }
    }
  } else {
    starts[support[0]] = 0;
  }
  if (i != len - 1) {

    if (support[threadIdx.x] != support[threadIdx.x + 1]) {
      ends[support[threadIdx.x]] = i;
    }
  } else {
    ends[support[threadIdx.x]] = len - 1;
  }
}

__global__ void findCellStart(int *keys, int len, int *starts) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i > 0) {
    if (keys[i] != keys[i - 1]) {
      starts[keys[i]] = i;
    }
  } else {
    starts[keys[0]] = 0;
  }
}

__global__ void distsQmin(float *Qx, float *Qy, float *Qz, float *Cx, float *Cy,
                          float *Cz, int lenghtC, float *x, float *y,
                          float *z) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float minq = 100;
  float dist;
  float tempx, tempy, tempz;
  for (int j = 0; j < lenghtC; j++) {
    dist = (Cx[j] - Qx[i]) * (Cx[j] - Qx[i]) +
           (Cy[j] - Qy[i]) * (Cy[j] - Qy[i]) +
           (Cz[j] - Qz[i]) * (Cz[j] - Qz[i]);
    dist = sqrt(dist);
    if (dist < minq) {
      minq = dist;
      tempx = Cx[j];
      tempy = Cy[j];
      tempz = Cz[j];
    }
  }
  x[i] = tempx;
  y[i] = tempy;
  z[i] = tempz;
}

__global__ void gridval(int *starts, int *ends, float *Cx, float *Cy, float *Cz,
                        int d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int digit1 = Cx[i] * d;
  int digit2 = Cy[i] * d;
  int digit3 = Cz[i] * d;
  int key = d * d * digit1 + d * digit2 + digit3;
  if (i > ends[key] || i < starts[key]) {
    printf("Error in grid construction \n");
  }
  if (i > 0) {
    digit1 = Cx[i - 1] * d;
    digit2 = Cy[i - 1] * d;
    digit3 = Cz[i - 1] * d;
    int key2 = d * d * digit1 + d * digit2 + digit3;
    if (key2 > key) {
      printf("Error not sorted\n");
    }
  }
}

__device__ void searchCell(float pointx, float pointy, float pointz, int start,
                           int end, float *Cx, float *Cy, float *Cz,
                           float *minq, float *x, float *y, float *z) {
  float dist;
  if (start >= 0) {
    for (int j = start; j <= end; j++) {
      dist = (Cx[j] - pointx) * (Cx[j] - pointx) +
             (Cy[j] - pointy) * (Cy[j] - pointy) +
             (Cz[j] - pointz) * (Cz[j] - pointz);
      dist = sqrt(dist);
      if (dist < *minq) {
        *minq = dist;
        *x = Cx[j];
        *y = Cy[j];
        *z = Cz[j];
      }
    }
  }
}

__device__ float findBoarderdistance(float pointx, float pointy, float pointz,
                                     int d, int s, int digit1, int digit2,
                                     int digit3) {
  float bdist = 100;

  if (digit1 + s < d - 1) {
    bdist = -pointx + (float)(digit1 + s + 1) / d;
  }
  if (digit2 + s < d - 1) {
    bdist = min(-pointy + (float)(digit2 + s + 1) / d, bdist);
  }
  if (digit3 + s < d - 1) {
    bdist = min(-pointz + (float)(digit3 + s + 1) / d, bdist);
  }
  if (digit1 - s > 0) {
    bdist = min(+pointx + (float)(-digit1 + s) / d, bdist);
  }
  if (digit2 - s > 0) {
    bdist = min(+pointy + (float)(-digit2 + s) / d, bdist);
  }
  if (digit3 - s > 0) {
    bdist = min(+pointz + (float)(-digit3 + s) / d, bdist);
  }
  return bdist;
}
__global__ void searchGrid(float *Qx, float *Qy, float *Qz, int *starts,
                           int *ends, float *Cx, float *Cy, float *Cz,
                           float *Resx, float *Resy, float *Resz, int d,
                           int lenghtQ, int *startsQ, int max) {

  for (int t = blockIdx.x; t < d * d * d; t = t + gridDim.x) {
    if (startsQ[t] >= 0) {

      extern __shared__ float buf[];
      float *Cxshared = buf;
      float *Cyshared = (float *)&Cxshared[-starts[t] + ends[t] + 1];
      float *Czshared = (float *)&Cyshared[-starts[t] + ends[t] + 1];

      for (int i = starts[t] + threadIdx.x; i <= ends[t]; i = i + blockDim.x) {
        Cxshared[i - starts[t]] = Cx[i];
        Cyshared[i - starts[t]] = Cy[i];
        Czshared[i - starts[t]] = Cz[i];
      }
      __syncthreads();

      int i = startsQ[t] + threadIdx.x;

      while (1) {
        if (i >= lenghtQ) {
          break;
        }

        float pointx = Qx[i];
        float pointy = Qy[i];
        float pointz = Qz[i];
        int digit1 = pointx * d;
        int digit2 = pointy * d;
        int digit3 = pointz * d;
        int key = d * d * digit1 + d * digit2 + digit3;

        if (key != t) {
          break;
        }
        float x = 0, y = 0, z = 0;
        float minq = 100;
        int s = 0;
        float bdist = findBoarderdistance(pointx, pointy, pointz, d, s, digit1,
                                          digit2, digit3);

        float dist;
        int digx, digy, digz;
        if (starts[key] >= 0) {
          for (int j = 0; j < ends[key] - starts[key] + 1; j++) {
            dist = (Cxshared[j] - pointx) * (Cxshared[j] - pointx) +
                   (Cyshared[j] - pointy) * (Cyshared[j] - pointy) +
                   (Czshared[j] - pointz) * (Czshared[j] - pointz);
            dist = sqrt(dist);
            if (dist < minq) {
              minq = dist;
              x = Cxshared[j];
              y = Cyshared[j];
              z = Czshared[j];
            }
          }
        }

        while (1) {
          s++;
          if (minq < bdist) {
            break;
          }
          for (digx = digit1 - s; digx <= digit1 + s; digx++) {
            for (digy = digit2 - s; digy <= digit2 + s; digy++) {
              for (digz = digit3 - s; digz <= digit3 + s; digz++) {
                if (digy == digit2 - s || digy == digit2 + s ||
                    digz == digit3 - s || digz == digit3 + s ||
                    digx == digit1 - s || digx == digit1 + s) {
                  key = d * d * digx + d * digy + digz;
                  if (key >= 0 && key < d * d * d) {
                    searchCell(pointx, pointy, pointz, starts[key], ends[key],
                               Cx, Cy, Cz, &minq, &x, &y, &z);
                  }
                }
              }
            }
          }
          bdist = findBoarderdistance(pointx, pointy, pointz, d, s, digit1,
                                      digit2, digit3);
        }

        Resx[i] = x;
        Resy[i] = y;
        Resz[i] = z;
        i = i + blockDim.x;
      }
    }
  }
}

__global__ void find_maximum_kernel(int *starts, int *ends, int *max,
                                    int *mutex, int n) {
  unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int stride = gridDim.x * blockDim.x;
  unsigned int offset = 0;

  extern __shared__ int cache[];

  int temp = -1.0;
  while (index + offset < n) {
    if (temp < -starts[index + offset] + ends[index + offset] + 1) {
      temp = -starts[index + offset] + ends[index + offset] + 1;
    }
    offset += stride;
  }

  cache[threadIdx.x] = temp;

  __syncthreads();

  // reduction
  unsigned int i = blockDim.x / 2;
  while (i != 0) {
    if (threadIdx.x < i) {
      if (cache[threadIdx.x] < cache[threadIdx.x + i]) {
        cache[threadIdx.x] = cache[threadIdx.x + i];
      }
    }

    __syncthreads();
    i /= 2;
  }

  if (threadIdx.x == 0) {
    while (atomicCAS(mutex, 0, 1) != 0)
      ; // lock
    if (*max < cache[0]) {
      *max = cache[0];
    }
    atomicExch(mutex, 0); // unlock
  }
}

void init_rand_points(float *p, int n) {
  int i;

  for (i = 0; i < n; i++) {
    p[i] = (float)(rand() - 1000) / (float)RAND_MAX;
  }
}
__global__ void checkmax(int max, int *starts, int *ends) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (ends[i] - starts[i] + 1 > max) {

    printf("Error Didin't find max\n");
  }
}

int main(int argc, char **argv) {

  if (argc != 3) {
    printf("Enter size of the set and queries as arguments \n");
    exit(0);
  }
  srand(time(NULL));

  int lenghtC = 1 << atoi(argv[1]);
  int lenghtQ = 1 << atoi(argv[2]);
  int d = 1 << 6;
  int numblocks = 1 << 10;
  int threadsPerBlock = 32;

  printf("Size of set 2 to %d  size of quiry set 2 to %d grid dimentions %d x "
         "%d x %d\n",
         atoi(argv[1]), atoi(argv[2]),d,d,d);
  printf("%d Threadblocks with %d threads per block \n", numblocks, threadsPerBlock );


  float *Csx, *Csy, *Csz, *Qsx, *Qsy, *Qsz;

  Csx = (float *)malloc(sizeof(float) * lenghtC);
  Csy = (float *)malloc(sizeof(float) * lenghtC);
  Csz = (float *)malloc(sizeof(float) * lenghtC);
  Qsx = (float *)malloc(sizeof(float) * lenghtQ);
  Qsy = (float *)malloc(sizeof(float) * lenghtQ);
  Qsz = (float *)malloc(sizeof(float) * lenghtQ);

  init_rand_points(Csx, lenghtC);
  init_rand_points(Csy, lenghtC);
  init_rand_points(Csz, lenghtC);
  init_rand_points(Qsx, lenghtQ);
  init_rand_points(Qsy, lenghtQ);
  init_rand_points(Qsz, lenghtQ);

  /*Allocate space in device memory*/
  float *Cx, *Cy, *Cz, *Qx, *Qy, *Qz;
  float *Resx, *Resy, *Resz;

  CUDA_CALL(hipMalloc(&Cx, lenghtC * sizeof(float)));
  CUDA_CALL(hipMalloc(&Cy, lenghtC * sizeof(float)));
  CUDA_CALL(hipMalloc(&Cz, lenghtC * sizeof(float)));
  CUDA_CALL(hipMalloc(&Qx, lenghtQ * sizeof(float)));
  CUDA_CALL(hipMalloc(&Qy, lenghtQ * sizeof(float)));
  CUDA_CALL(hipMalloc(&Qz, lenghtQ * sizeof(float)));

  hipMemcpy(Cx, Csx, lenghtC * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Cy, Csy, lenghtC * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Cz, Csz, lenghtC * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(Qx, Qsx, lenghtQ * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Qy, Qsy, lenghtQ * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Qz, Qsz, lenghtQ * sizeof(float), hipMemcpyHostToDevice);

  /*Make pointers so we can use the thrust libraty*/
  thrust ::device_ptr<float> Cx_ptr(Cx);
  thrust ::device_ptr<float> Cy_ptr(Cy);
  thrust ::device_ptr<float> Cz_ptr(Cz);
  thrust ::device_ptr<float> Qx_ptr(Qx);
  thrust ::device_ptr<float> Qy_ptr(Qy);
  thrust ::device_ptr<float> Qz_ptr(Qz);

  /*Count time for grid creation*/
  float gridMakeTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  /*Find the grid node for each point in C*/
  int *keysC;
  CUDA_CALL(hipMalloc(&keysC, lenghtC * sizeof(int)));

  generatekey<<<lenghtC / threadsPerBlock, threadsPerBlock>>>(
      Cx, Cy, Cz, lenghtC, keysC, d);
  hipDeviceSynchronize();

  /*Sort by grid node*/
  thrust ::device_ptr<int> kc(keysC);
  thrust ::stable_sort_by_key(
      kc, kc + lenghtC, make_zip_iterator(make_tuple(Cx_ptr, Cy_ptr, Cz_ptr)));

  /*Find where its key starts and where it ends in the sorted array*/
  int *starts;
  CUDA_CALL(hipMalloc(&starts, d * d * d * (sizeof(int))));
  CUDA_CALL(hipMemset(starts, -1, d * d * d * (sizeof(int))));
  int *ends;
  CUDA_CALL(hipMalloc(&ends, d * d * d * (sizeof(int))));
  CUDA_CALL(hipMemset(ends, -1, d * d * d * (sizeof(int))));
  findCellStartends<<<lenghtC / threadsPerBlock, threadsPerBlock,
                      (threadsPerBlock + 2) * sizeof(int)>>>(keysC, lenghtC,
                                                             starts, ends);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gridMakeTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("Time for grid creation   %f  ms \n", gridMakeTime);

  /*Count sorting time for Q ie Putting Q in the grid*/
  float sortTime;

  hipEvent_t startsort, stopsort;
  hipEventCreate(&startsort);
  hipEventCreate(&stopsort);
  hipEventRecord(startsort, 0);

  int *keysQ;
  CUDA_CALL(hipMalloc(&keysQ, lenghtQ * sizeof(int)));
  generatekey<<<lenghtQ / threadsPerBlock, threadsPerBlock>>>(
      Qx, Qy, Qz, lenghtQ, keysQ, d);
  hipDeviceSynchronize();
  thrust ::device_ptr<int> kq(keysQ);
  thrust ::stable_sort_by_key(
      kq, kq + lenghtQ, make_zip_iterator(make_tuple(Qx_ptr, Qy_ptr, Qz_ptr)));
  hipDeviceSynchronize();
  int *startsQ;
  CUDA_CALL(hipMalloc(&startsQ, d * d * d * (sizeof(int))));
  CUDA_CALL(hipMemset(startsQ, -1, d * d * d * (sizeof(int))));
  findCellStart<<<lenghtQ / threadsPerBlock, threadsPerBlock>>>(keysQ, lenghtQ,
                                                                startsQ);

  hipEventRecord(stopsort, 0);
  hipEventSynchronize(stopsort);
  hipEventElapsedTime(&sortTime, startsort, stopsort);
  hipEventDestroy(startsort);
  hipEventDestroy(stopsort);

  printf("Sorting Q time  %f  ms \n", sortTime);

  gridval<<<lenghtC / threadsPerBlock, threadsPerBlock>>>(starts, ends, Cx, Cy,
                                                          Cz, d);

  CUDA_CALL(hipMalloc(&Resx, lenghtQ * sizeof(float)));
  CUDA_CALL(hipMalloc(&Resy, lenghtQ * sizeof(float)));
  CUDA_CALL(hipMalloc(&Resz, lenghtQ * sizeof(float)));

  float maxtime;
  hipEvent_t startmax, stopmax;
  hipEventCreate(&startmax);
  hipEventCreate(&stopmax);
  hipEventRecord(startmax, 0);

  int *mutex;
  CUDA_CALL(hipMalloc(&mutex, sizeof(int)));
  CUDA_CALL(hipMemset(mutex, 0, (sizeof(int))));
  int *d_max;
  CUDA_CALL(hipMalloc(&d_max, sizeof(int)));
  CUDA_CALL(hipMemset(d_max, -1, (sizeof(int))));
  int h_max;
  find_maximum_kernel<<<numblocks, threadsPerBlock,
                        threadsPerBlock * sizeof(int)>>>(starts, ends, d_max,
                                                         mutex, d * d * d);
  hipDeviceSynchronize();
  hipMemcpy(&h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);

  hipEventRecord(stopmax, 0);
  hipEventSynchronize(stopmax);
  hipEventElapsedTime(&maxtime, startmax, stopmax);
  hipEventDestroy(startmax);
  hipEventDestroy(stopmax);
  printf("Finding  Maximum number of points in a shell time  %f ms  \n", maxtime);

  checkmax<<<d * d * d / threadsPerBlock, threadsPerBlock>>>(h_max, starts,
                                                             ends);
  hipDeviceSynchronize();
  float elapsedTime;
  hipEvent_t startse, stopse;
  hipEventCreate(&startse);
  hipEventCreate(&stopse);
  hipEventRecord(startse, 0);

  searchGrid<<<numblocks, threadsPerBlock, 3 * h_max * sizeof(float)>>>(
      Qx, Qy, Qz, starts, ends, Cx, Cy, Cz, Resx, Resy, Resz, d, lenghtQ,
      startsQ, h_max);

  hipEventRecord(stopse, 0);
  hipEventSynchronize(stopse);
  hipEventElapsedTime(&elapsedTime, startse, stopse);
  hipEventDestroy(startse);
  hipEventDestroy(stopse);
  printf("Search Time   %f ms  \n", elapsedTime);

  float *x;
  float *z;
  float *y;

  CUDA_CALL(hipMalloc(&x, lenghtQ * sizeof(float)));
  CUDA_CALL(hipMalloc(&y, lenghtQ * sizeof(float)));
  CUDA_CALL(hipMalloc(&z, lenghtQ * sizeof(float)));

  /*    Validation  */


  distsQmin<<<lenghtQ / threadsPerBlock, threadsPerBlock>>>(
      Qx, Qy, Qz, Cx, Cy, Cz, lenghtC, x, y, z);

  float *gridx, *gridy, *gridz, *minx, *miny, *minz;
  gridx = (float *)malloc(lenghtQ * sizeof(float));
  gridy = (float *)malloc(lenghtQ * sizeof(float));
  gridz = (float *)malloc(lenghtQ * sizeof(float));
  minx = (float *)malloc(lenghtQ * sizeof(float));
  miny = (float *)malloc(lenghtQ * sizeof(float));
  minz = (float *)malloc(lenghtQ * sizeof(float));
  hipMemcpy(gridx, Resx, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(gridy, Resy, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(gridz, Resz, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(minx, x, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(miny, y, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(minz, z, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);

  float *Qhx, *Qhy, *Qhz;
  Qhx = (float *)malloc(lenghtQ * sizeof(float));
  Qhy = (float *)malloc(lenghtQ * sizeof(float));
  Qhz = (float *)malloc(lenghtQ * sizeof(float));
  hipMemcpy(Qhx, Qx, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Qhy, Qy, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Qhz, Qz, lenghtQ * sizeof(float), hipMemcpyDeviceToHost);

  int s = 0;
  int c = 0;
  for (int i = 0; i < lenghtQ; i++) {
    if (minx[i] != gridx[i] || miny[i] != gridy[i] || minz[i] != gridz[i]) {
      s++;
    } else {
      c++;
    }
  }
  printf("Wrong number of points %d ", s);
  printf("Ritght number of points  %d \n", c);

  CUDA_CALL(hipFree(Cx));
  CUDA_CALL(hipFree(Cy));
  CUDA_CALL(hipFree(Cz));
  CUDA_CALL(hipFree(Qx));
  CUDA_CALL(hipFree(Qy));
  CUDA_CALL(hipFree(Qz));
  CUDA_CALL(hipFree(Resx));
  CUDA_CALL(hipFree(Resy));
  CUDA_CALL(hipFree(Resz));
  CUDA_CALL(hipFree(keysC));
  CUDA_CALL(hipFree(keysQ));
  CUDA_CALL(hipFree(ends));
  CUDA_CALL(hipFree(starts));
  CUDA_CALL(hipFree(startsQ));
  CUDA_CALL(hipFree(x));
  CUDA_CALL(hipFree(y));
  CUDA_CALL(hipFree(z));

  return 0;
}
